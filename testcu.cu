#include <iostream>
#include <cmath>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <omp.h>
// CUDA 内核函数，执行矩阵乘法
__global__ void sgemm_kernel(const int64_t N, const int64_t M, const int64_t K, float *A, float *B, float *C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < M) {
        float sum = 0.0f;
        for (int64_t k = 0; k < K; ++k) {
            sum += A[row * K + k] * B[col * K + k];
        }
        C[row * M + col] = sum;
    }
}

// 封装的 CPU 函数，负责内存管理和内核调用
void Sgemm(const int64_t N, const int64_t M, const int64_t K, float *A, float *B, float *C) {
    float *d_A, *d_B, *d_C;
    size_t size_A = N * K * sizeof(float);
    size_t size_B = M * K * sizeof(float);
    size_t size_C = N * M * sizeof(float);

    // 分配 GPU 内存
    hipMalloc((void**)&d_A, size_A);
    hipMalloc((void**)&d_B, size_B);
    hipMalloc((void**)&d_C, size_C);

    // 将数据从 CPU 复制到 GPU
    hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size_B, hipMemcpyHostToDevice);

    // 定义线程块和网格的维度
    dim3 blockSize(16, 16);
    dim3 gridSize((M + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);

    // 调用 CUDA 内核
    sgemm_kernel<<<gridSize, blockSize>>>(N, M, K, d_A, d_B, d_C);

    // 将结果从 GPU 复制回 CPU
    hipMemcpy(C, d_C, size_C, hipMemcpyDeviceToHost);

    // 释放 GPU 内存
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
} 

void sgemm(const int64_t N, const int64_t M, const int64_t K, float *A, float *B, float *C) {
	float sum ;
    #pragma omp parallel for collapse(2)
    for (int64_t n = 0; n < N; ++n) {
        for (int64_t m = 0; m < M; ++m) {
            sum = 0;
            for (int64_t k = 0; k < K; ++k) {
                sum += A[n * K + k] * B[m * K + k];
            }
            C[n * M + m] = sum;
        }
    }
}

int main() {
    const int64_t N = 100;
    const int64_t M = 100;
    const int64_t K = 100;

    float *A = new float[N * K];
    float *B = new float[K * M];
    float *C_cuda = new float[N * M];
    float *C_omp = new float[N * M];

    // 初始化矩阵 A 和 B
    for (int64_t i = 0; i < N * K; ++i) {
        A[i] = static_cast<float>(rand()) / RAND_MAX;
    }
    for (int64_t i = 0; i < K * M; ++i) {
        B[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // 使用 CUDA 计算矩阵乘法
    Sgemm(N, M, K, A, B, C_cuda);

    // 使用 OpenMP 计算矩阵乘法
    sgemm(N, M, K, A, B, C_omp);

    // 计算最大绝对误差
    float max_error = 0.0f;
    for (int64_t i = 0; i < N * M; ++i) {
        float error = std::abs(C_cuda[i] - C_omp[i]);
        if (error > max_error) {
            max_error = error;
        }
    }

    std::cout << "Maximum absolute error: " << max_error << std::endl;

    // 释放内存
    delete[] A;
    delete[] B;
    delete[] C_cuda;
    delete[] C_omp;

    return 0;
}
