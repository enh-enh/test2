#include <iostream>
#include <cmath>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <hipblas.h>

void Sgemm(const int64_t N, const int64_t M, const int64_t K, float *A, float *B, float *C) {
    hipblasHandle_t handle;
    float *d_A, *d_B, *d_C;
    size_t size_A = N * K * sizeof(float);
    size_t size_B = M * K * sizeof(float);
    size_t size_C = N * M * sizeof(float);

    hipMalloc((void**)&d_A, size_A);
    hipMalloc((void**)&d_B, size_B);
    hipMalloc((void**)&d_C, size_C);

    hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size_B, hipMemcpyHostToDevice);

    hipblasCreate(&handle);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                static_cast<int>(M), static_cast<int>(N), static_cast<int>(K), 
                &alpha, d_B, static_cast<int>(M), 
                d_A, static_cast<int>(K), 
                &beta, d_C, static_cast<int>(M));

    hipMemcpy(C, d_C, size_C, hipMemcpyDeviceToHost);

    hipblasDestroy(handle);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
