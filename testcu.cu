#include <iostream>
#include <cmath>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <omp.h>

// CUDA 规约求和的线程块大小
const int TILE_SIZE = 16;

__global__ void sgemm_kernel(const int64_t N, const int64_t M, const int64_t K, float *A, float *B, float *C) {
    __shared__ float sharedA[TILE_SIZE][TILE_SIZE];
    __shared__ float sharedB[TILE_SIZE][TILE_SIZE];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * blockDim.y + ty;
    int col = bx * blockDim.x + tx;

    float sum = 0.0f;

    for (int t = 0; t < (K + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        if (row < N && t * TILE_SIZE + tx < K) {
            sharedA[ty][tx] = A[row * K + t * TILE_SIZE + tx];
        } else {
            sharedA[ty][tx] = 0.0f;
        }

        if (col < M && t * TILE_SIZE + ty < K) {
            sharedB[ty][tx] = B[(t * TILE_SIZE + ty) * M + col];
        } else {
            sharedB[ty][tx] = 0.0f;
        }

        __syncthreads();

        for (int k = 0; k < TILE_SIZE; ++k) {
            sum += sharedA[ty][k] * sharedB[k][tx];
        }

        __syncthreads();
    }

    if (row < N && col < M) {
        C[row * M + col] = sum;
    }
}

void Sgemm(const int64_t N, const int64_t M, const int64_t K, float *A, float *B, float *C) {
    float *d_A, *d_B, *d_C;
    size_t size_A = N * K * sizeof(float);
    size_t size_B = K * M * sizeof(float);
    size_t size_C = N * M * sizeof(float);

    hipMalloc((void**)&d_A, size_A);
    hipMalloc((void**)&d_B, size_B);
    hipMalloc((void**)&d_C, size_C);

    hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size_B, hipMemcpyHostToDevice);

    dim3 blockSize(TILE_SIZE, TILE_SIZE);
    dim3 gridSize((M + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);

    sgemm_kernel<<<gridSize, blockSize>>>(N, M, K, d_A, d_B, d_C);

    hipMemcpy(C, d_C, size_C, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

void sgemm(const int64_t N, const int64_t M, const int64_t K, float *A, float *B, float *C) {
    float sum;
    #pragma omp parallel for collapse(2)
    for (int64_t n = 0; n < N; ++n) {
        for (int64_t m = 0; m < M; ++m) {
            sum = 0;
            for (int64_t k = 0; k < K; ++k) {
                sum += A[n * K + k] * B[k * M + m];
            }
            C[n * M + m] = sum;
        }
    }
}

int main() {
    const int64_t N = 100;
    const int64_t M = 100;
    const int64_t K = 100;

    float *A = new float[N * K];
    float *B = new float[K * M];
    float *C_cuda = new float[N * M];
    float *C_omp = new float[N * M];

    // 初始化矩阵 A 和 B
    for (int64_t i = 0; i < N * K; ++i) {
        A[i] = static_cast<float>(rand()) / RAND_MAX;
    }
    for (int64_t i = 0; i < K * M; ++i) {
        B[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // 使用 CUDA 计算矩阵乘法
    Sgemm(N, M, K, A, B, C_cuda);

    // 使用 OpenMP 计算矩阵乘法
    sgemm(N, M, K, A, B, C_omp);

    // 计算最大绝对误差
    float max_error = 0.0f;
    for (int64_t i = 0; i < N * M; ++i) {
        float error = std::abs(C_cuda[i] - C_omp[i]);
        if (error > max_error) {
            max_error = error;
        }
    }

    std::cout << "Maximum absolute error: " << max_error << std::endl;

    // 释放内存
    delete[] A;
    delete[] B;
    delete[] C_cuda;
    delete[] C_omp;

    return 0;
}
