#include <iostream>
#include "hip/hip_runtime.h"

__global__ void sgemm_kernel(const int64_t N, const int64_t M, const int64_t K, float *A, float *B, float *C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int n = idx / M;
    int m = idx % M;

    if (n < N && m < M) {
        float sum = 0;
        for (int64_t k = 0; k < K; ++k) {
            sum += A[n * K + k] * B[m * K + k];
        }
        C[n * M + m] = sum;
    }
}

void Sgemm(const int64_t N, const int64_t M, const int64_t K, float *A, float *B, float *C) {
    float *d_A, *d_B, *d_C;
    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&d_A, N * K * sizeof(float));
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMalloc d_A failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return;
    }
    cudaStatus = hipMalloc((void**)&d_B, M * K * sizeof(float));
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMalloc d_B failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_A);
        return;
    }
    cudaStatus = hipMalloc((void**)&d_C, N * M * sizeof(float));
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMalloc d_C failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_A);
        hipFree(d_B);
        return;
    }

    cudaStatus = hipMemcpy(d_A, A, N * K * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMemcpy h_A -> d_A failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        return;
    }
    cudaStatus = hipMemcpy(d_B, B, M * K * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMemcpy h_B -> d_B failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        return;
    }

    int blockSize = 256;
    int gridSize = (N * M + blockSize - 1) / blockSize;

    sgemm_kernel<<<gridSize, blockSize>>>(N, M, K, d_A, d_B, d_C);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "Kernel launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        return;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipDeviceSynchronize failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        return;
    }

    cudaStatus = hipMemcpy(C, d_C, N * M * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMemcpy d_C -> h_C failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        return;
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
